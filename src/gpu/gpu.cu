#include "../include/gpu/gpu.h"

#include <cstdio>

namespace ginf {
	template <typename T>
	T *Gpu::createRawMat(Matrix<T> *hostM) {
		T *dMat = NULL;

		// Allocate memory on device
		GINF_SAFE_CALL(hipMalloc((void **)&dMat, sizeof(T) * hostM->getTotalSize()));

		// Copy matrix to device
		GINF_SAFE_CALL(hipMemcpy(dMat, hostM->data, sizeof(T) * hostM->getTotalSize(), hipMemcpyHostToDevice));
		
		return dMat;
	}
	
	template <typename T>
	T *Gpu::createRawMat(int length) {
		T *dMat = NULL;

		// Allocate memory on device
		GINF_SAFE_CALL(hipMalloc((void **)&dMat, sizeof(T) * length));

		// Set all the elements to 0
		GINF_SAFE_CALL(hipMemset(dMat, 0, sizeof(T) * length));
		
		return dMat;
	}

	template <typename T>
	GpuGrid<T> *Gpu::createGrid(Grid<T> *hostG) {
		GpuGrid<T> *dGrid = NULL;

		// We'll need to use a temporary GpuGrid to hold device addresses of
		// the cost matrices
		GpuGrid<T> tempGrid;
		tempGrid.smModel = hostG->getSmModel();
		tempGrid.dimDt = MatDim(hostG->getWidth(), hostG->getHeight(), hostG->getNumLabels());
		tempGrid.dimSm = MatDim(hostG->getNumLabels(), hostG->getNumLabels());

		// Allocate grid on device
		// Use the tempGrid to hold device addresses for data and smoothness cost
		// matrices, and then copy tempGrid to our actual dGrid
		GINF_SAFE_CALL(hipMalloc((void **)&(tempGrid.dtCosts),
			sizeof(T) * hostG->getDataCosts()->getTotalSize()));
		GINF_SAFE_CALL(hipMalloc((void **)&(tempGrid.smCosts),
			sizeof(T) * hostG->getSmCosts()->getTotalSize()));
		GINF_SAFE_CALL(hipMalloc((void **)&dGrid, sizeof(GpuGrid<T>)));

		// Copy grid to device
		GINF_SAFE_CALL(hipMemcpy(tempGrid.dtCosts, hostG->getDataCosts()->data,
			sizeof(T) * hostG->getDataCosts()->getTotalSize(), hipMemcpyHostToDevice));
		GINF_SAFE_CALL(hipMemcpy(tempGrid.smCosts, hostG->getSmCosts()->data,
			sizeof(T) * hostG->getSmCosts()->getTotalSize(), hipMemcpyHostToDevice));
		GINF_SAFE_CALL(hipMemcpy(dGrid, &tempGrid, sizeof(GpuGrid<T>), hipMemcpyHostToDevice));

		return dGrid;
	}
	
	template <typename T>
	void Gpu::copyRawMat(T *deviceM, Matrix<T> *hostM) {
		GINF_SAFE_CALL(hipMemcpy(deviceM, hostM->data, sizeof(T) * hostM->getTotalSize(), hipMemcpyHostToDevice));
	}
	
	template <typename T>
	void Gpu::copyRawMat(Matrix<T> *hostM, T *deviceM) {
		GINF_SAFE_CALL(hipMemcpy(hostM->data, deviceM, sizeof(T) * hostM->getTotalSize(), hipMemcpyDeviceToHost));
	}
	
	template <typename T>
	void Gpu::copyRawMat(T *deviceDst, T *deviceSrc, int len) {
		GINF_SAFE_CALL(hipMemcpy(deviceDst, deviceSrc, sizeof(T) * len, hipMemcpyDeviceToDevice));
	}
	
	template <typename T>
	void Gpu::free(T *dPtr) {
		GINF_SAFE_CALL(hipFree(dPtr));
	}
	
	template <typename T>
	void Gpu::free(GpuGrid<T> *dGrid) {
		GINF_SAFE_CALL(hipFree(dGrid));
	}

	void cudaAssert(const hipError_t err, const char *file, const int line) {
	    if (hipSuccess != err) {
			fprintf(stderr, "Cuda error in file '%s' in line %i : %s.\n",
			        file, line, hipGetErrorString(err) );
			exit(1);
	    }
	}
}
